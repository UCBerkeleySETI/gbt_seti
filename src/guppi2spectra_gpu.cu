#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "guppi2spectra_gpu.h"
#include <stdio.h>

extern "C" void explode_wrapper(unsigned char *channelbufferd, hipfftComplex * voltages, int veclen);
extern "C" void detect_wrapper(hipfftComplex * voltages, int veclen, int fftlen, float *bandpassd, float *spectrumd);
extern "C" void setQuant(float *lut);
extern "C" void normalize_wrapper(float * tree_dedopplerd_pntr, float *mean, float *stddev, int tdwidth);
extern "C" void vecdivide_wrapper(float * spectrumd, float * divisord, int tdwidth);
extern "C" void explode8_wrapper(char *channelbufferd, hipfftComplex * voltages, int veclen);
extern "C" void explode8init_wrapper(char *channelbufferd, int veclen);
extern "C" void explode8simple_wrapper(char *channelbufferd, hipfftComplex * voltages, int veclen);


__constant__ float gpu_qlut[4];
__constant__ float meand;
__constant__ float stddevd;

texture<char, hipTextureType1D, hipReadModeNormalizedFloat> char_tex;


__global__ void explode8(char *channelbuffer, hipfftComplex * voltages, int veclen) {

int tid = threadIdx.x + blockIdx.x * blockDim.x;

	 if(tid < veclen) {	 
		  voltages[tid].x = tex1Dfetch(char_tex, channelbuffer[4*tid]); 
		  voltages[tid].y = tex1Dfetch(char_tex, channelbuffer[4*tid + 1]);
		  voltages[veclen + tid].x = tex1Dfetch(char_tex, channelbuffer[4*tid + 2]); 
		  voltages[veclen + tid].y = tex1Dfetch(char_tex, channelbuffer[4*tid + 3]);
	 }
	 
}

__global__ void explode8simple(char *channelbuffer, hipfftComplex * voltages, int veclen) {

int tid = threadIdx.x + blockIdx.x * blockDim.x;

	 if(tid < veclen) {	 
		  voltages[tid].x = (float) channelbuffer[4*tid]; 
		  voltages[tid].y = (float) channelbuffer[4*tid + 1];
		  voltages[veclen + tid].x = (float) channelbuffer[4*tid + 2]; 
		  voltages[veclen + tid].y = (float) channelbuffer[4*tid + 3];
	 }

}

__global__ void explode(unsigned char *channelbuffer, hipfftComplex * voltages, int veclen) {

int tid = threadIdx.x + blockIdx.x * blockDim.x;

//float lookup[4];
//lookup[0] = 3.3358750;
//lookup[1] = 1.0;
//lookup[2] = -1.0;
//lookup[3] = -3.3358750;

	 if(tid < veclen) {	 
		  voltages[tid].x = gpu_qlut[(channelbuffer[tid] >> (0 * 2) & 1) +  (2 * (channelbuffer[tid] >> (0 * 2 + 1) & 1))];
		  voltages[tid].y = gpu_qlut[(channelbuffer[tid] >> (1 * 2) & 1) +  (2 * (channelbuffer[tid] >> (1 * 2 + 1) & 1))];
		  voltages[veclen + tid].x = gpu_qlut[(channelbuffer[tid] >> (2 * 2) & 1) +  (2 * (channelbuffer[tid] >> (2 * 2 + 1) & 1))];
		  voltages[veclen + tid].y = gpu_qlut[(channelbuffer[tid] >> (3 * 2) & 1) +  (2 * (channelbuffer[tid] >> (3 * 2 + 1) & 1))];
	 }
	 
}


__global__ void detect(hipfftComplex * voltages, int veclen, int fftlen, float * bandpassd, float * spectrumd) {

int tid = threadIdx.x + blockIdx.x * blockDim.x;
int indx = tid - (tid%fftlen) + (tid + fftlen/2)%fftlen;

//73 - (73%16) + (73 + 8)%16

	 if(tid < veclen) {
		  //spectrumd[tid] = ((voltages[((tid+fftlen/2)%fftlen)].x * voltages[((tid+fftlen/2)%fftlen)].x) + (voltages[((tid+fftlen/2)%fftlen)].y * voltages[((tid+fftlen/2)%fftlen)].y) + (voltages[fftlen + ((tid+fftlen/2)%fftlen)].x * voltages[fftlen + ((tid+fftlen/2)%fftlen)].x)+ (voltages[fftlen + ((tid+fftlen/2)%fftlen)].y * voltages[fftlen + ((tid+fftlen/2)%fftlen)].y))/bandpassd[tid];	 		  
		  spectrumd[tid] = spectrumd[tid] + ((voltages[indx].x * voltages[indx].x) + (voltages[indx].y * voltages[indx].y) + (voltages[veclen + indx].x * voltages[veclen + indx].x)+ (voltages[veclen + indx].y * voltages[veclen + indx].y));	 		  
	 }
}



__global__ void normalize(float * tree_dedopplerd_pntr, int tdwidth)  {

int tid = threadIdx.x + blockIdx.x * blockDim.x;

	 if(tid < tdwidth) { 
		tree_dedopplerd_pntr[tid] = (tree_dedopplerd_pntr[tid] - meand)/stddevd;     
	 }

}

__global__ void vecdivide(float * spectrumd, float * divisord, int tdwidth)  {

int tid = threadIdx.x + blockIdx.x * blockDim.x;

	 if(tid < tdwidth) { 
		spectrumd[tid] = spectrumd[tid]/divisord[tid];     
	 }

}



void explode_wrapper(unsigned char *channelbufferd, hipfftComplex * voltages, int veclen) {
	explode<<<veclen/1024,1024>>>(channelbufferd, voltages, veclen);
}

void detect_wrapper(hipfftComplex * voltages, int veclen, int fftlen, float *bandpassd, float *spectrumd) {
	detect<<<veclen/1024,1024>>>(voltages, veclen, fftlen, bandpassd, spectrumd);
}


//veclen is number of complex elements, so length of channelbufferd is 2 x veclen
void explode8_wrapper(char *channelbufferd, hipfftComplex * voltages, int veclen) {
	explode8<<<veclen/1024,1024>>>(channelbufferd, voltages, veclen);
}


//veclen is number of complex elements, so length of channelbufferd is 2 x veclen
void explode8simple_wrapper(char *channelbufferd, hipfftComplex * voltages, int veclen) {
	explode8simple<<<veclen/1024,1024>>>(channelbufferd, voltages, veclen);
}

void explode8init_wrapper(char *channelbufferd, int length) {
	hipBindTexture(0, char_tex, channelbufferd, length);
}




void setQuant(float *lut) {
#if CUDA_VERSION >= 4500
        fprintf(stderr, "loading lookuptable...%s\n", hipGetErrorString(hipMemcpyToSymbol(HIP_SYMBOL(gpu_qlut), lut, 16, 0, hipMemcpyHostToDevice)));
#else
        fprintf(stderr, "loading lookuptable...%s\n", hipGetErrorString(hipMemcpyToSymbol(HIP_SYMBOL("gpu_qlut"), lut, 16, 0, hipMemcpyHostToDevice)));
#endif

}

void normalize_wrapper(float * tree_dedopplerd_pntr, float *mean, float *stddev, int tdwidth) {
	
	hipMemcpyToSymbol(HIP_SYMBOL("meand"), mean, 4, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("stddevd"), stddev, 4, 0, hipMemcpyHostToDevice);

	normalize<<<(tdwidth+511)/512,512>>>(tree_dedopplerd_pntr, tdwidth);
}

void vecdivide_wrapper(float * spectrumd, float * divisord, int tdwidth) {
	
	vecdivide<<<(tdwidth+511)/512,512>>>(spectrumd, divisord, tdwidth);
}

